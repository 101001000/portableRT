#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <cstdio>
#include <cstdint>
#include "../include/portableRT/hitreg.hpp"

template<class... Tags>
struct Params        
{
    OptixTraversableHandle handle;
    float4*  origins;     
    float4*  directions;
    portableRT::HitReg<Tags...>*    results;    
};

template<class... Tags>
static __forceinline__ __device__
void raygen_body(const Params<Tags...>& params)
{
    const uint3 idx = optixGetLaunchIndex();
    const int i = idx.x;

    float4 o4 = params.origins[i];
    float4 d4 = params.directions[i];
    float3 origin = make_float3(o4.x, o4.y, o4.z);
    float3 direction = make_float3(d4.x, d4.y, d4.z);

    unsigned int p0, p1, p2, p3;

    optixTrace( params.handle,
                origin,
                direction,
                0.0f,              
                10000000.0f,           
                0.0f,     
                OptixVisibilityMask( 255 ),
                OPTIX_RAY_FLAG_NONE,
                0, 1, 0,           
                p0, p1, p2, p3 );
    params.results[i].t = __uint_as_float(p0);
    params.results[i].u = __uint_as_float(p1);
    params.results[i].v = __uint_as_float(p2);
    params.results[i].primitive_id = p3;
    params.results[i].valid = isfinite(__uint_as_float(p0));
    params.results[i].px = o4.x + __uint_as_float(p0) * d4.x;
    params.results[i].py = o4.y + __uint_as_float(p0) * d4.y;
    params.results[i].pz = o4.z + __uint_as_float(p0) * d4.z;
}

#define X(...) DEFINE_RAYGEN(__VA_ARGS__)
TAG_COMBOS
#undef X

extern "C" __global__ void __miss__ms(){
    optixSetPayload_0(__float_as_uint(INFINITY));
    //optixSetPayload_1(0xFFFFFFFFu);
}


extern "C" __global__ void __closesthit__ch() {
    float2 barycentric = optixGetTriangleBarycentrics();
    optixSetPayload_0(__float_as_uint(optixGetRayTmax()));
    optixSetPayload_1(__float_as_uint(barycentric.x));
    optixSetPayload_2(__float_as_uint(barycentric.y));
    optixSetPayload_3(optixGetPrimitiveIndex());
}